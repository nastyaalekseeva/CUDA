#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<iostream>
using namespace std;
int  main()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	printf("Device name:%s\n", deviceProp.name);
	printf("Multiprocessor count:%d\n", deviceProp.multiProcessorCount);
	printf("Total global memory:%u\n", deviceProp.totalGlobalMem);
	printf("Memory clock rate:%d\n", deviceProp.memoryClockRate);
	printf("Clock rate:%d\n", deviceProp.clockRate); 
	printf("Memory bus width:%d\n", deviceProp.memoryBusWidth);
	scanf("");
	system("pause");
}