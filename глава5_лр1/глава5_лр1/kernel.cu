#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#define n 1000
#define MAX_GRIDSIZE 1
__device__ float __expf(float x);
__global__ void expMass(float *A,float *x, int arraySize)
{
	int index = (blockIdx.y*MAX_GRIDSIZE + blockIdx.x)*blockDim.x + threadIdx.x;
	if (index < arraySize)
		A[index]=expf(x[index]);
}
int main()
{
	float a = 1, b = 5;
	float h = ((10-1)*1.0)/n;
	float x[n], S[n],arr[n];
	float *A, *dev_x;
	x[0] = a;
	for (int i = 1; i < n; i++)
	{
		x[i] = i*h;
		arr[i] = exp(x[i]);
	}

	int size = sizeof(float);
	hipMalloc((void**)&A, n*size);
	hipMalloc((void**)&dev_x, n*size);
	hipMemcpy(dev_x, &x, n*size, hipMemcpyHostToDevice);
	expMass << <1, n >> > (A, dev_x,n);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("GPUassert: %s %s %d\n", hipGetErrorString(err));
	}
	hipMemcpy(S, A, n*size, hipMemcpyDeviceToHost);
	for (int i = 1; i < n; i++)
	{
		printf("x=%f,  S=%f,  arr=%f,  err=%f \n", x[i], S[i], arr[i],abs(S[i] - arr[i]) / n);
	}
	getchar();
}