#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cmath>
#include <assert.h>
#define N 10000
void checkCUDAError(const char *msg) 
{ 
	hipError_t err = hipGetLastError();  
	if (hipSuccess != err) 
	{ 
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));   
		exit(EXIT_FAILURE); 
	} 
}
texture<float, 1, hipReadModeElementType> texRef;
__global__ void Integral( float *C, float h, int n)
{
		int j = blockIdx.x*blockDim.x + threadIdx.x;

		if (j < n)

		{
			float x = ((tex1Dfetch(texRef, j)+ (tex1Dfetch(texRef, j) + *h))) / 2;
			C[j]= h*sqrt(1 - x*x);
		}

}
#define  NUM_THREADS 100 
int main()
{
	int nBlocks = N/NUM_THREADS;
	float a = 0, b = 1;
	float h = ((b - a)*1.0) / N;
	float x[N], S[N];//CPU
	float *dev_S; float *dev_x, *dev_h;//GPU
	x[0] = a;
	for (int i = 1; i < N; i++)
	{
		x[i] = i*h;
	}
	int size = sizeof(float);
	int memSize = N * sizeof(float);
	hipMalloc((void**)&dev_S, memSize);
	hipMalloc((void**)&dev_x, memSize);
	hipMalloc((void**)&dev_h, size);
	hipMemcpy(dev_x, &x, memSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_h, &h, size, hipMemcpyHostToDevice);
	hipBindTexture(0, texRef, dev_x,memSize);
	checkCUDAError("bind");
	Integral << < nBlocks, NUM_THREADS >> >(dev_S, dev_h,N);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("GPUassert: %s %s %d\n", hipGetErrorString(err));
	}
	hipMemcpy(&S, dev_S,N*size, hipMemcpyDeviceToHost);
	float p = 0.0;
	for (int i = 0; i < N; i++)
	{
		p = p + S[i];
	}
	hipFree(dev_S);
	hipUnbindTexture(texRef);
	printf("Pi = %f ", 4 * p);
	getchar();
}
