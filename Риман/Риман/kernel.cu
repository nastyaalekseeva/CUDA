
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <cmath>
#define  n 10000
#define BLOCK 10
__global__ void Su(float *S_d, float *x)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	float q = 1.0;
	for (int j = 1; j <= *x; j++)
	{
		q = q*i;
	}
	S_d[i] =  1./q;	
}
int main()
{

	float host_x=2;//CPU
	float *dev_S;float *dev_x;//GPU
	float host_S[n];
	int size = sizeof(float);
	hipMalloc((void**)&dev_S, n*size);
	hipMalloc((void**)&dev_x, size);
	hipMemcpy(dev_S, &host_S, n*size, hipMemcpyHostToDevice);
	hipMemcpy(dev_x, &host_x, size, hipMemcpyHostToDevice);
	
	Su <<< BLOCK, n / BLOCK >>>(dev_S,dev_x);
	hipDeviceSynchronize();
	hipMemcpy(&host_S, dev_S, n*size, hipMemcpyDeviceToHost);
	float p = 0.0;
	for (int i = 1; i < n; i++)
	{
		p = p + host_S[i];
	}
	printf("S = %f ", p);
	hipFree(dev_S);
	hipFree(dev_x);
	
	getchar();
}
