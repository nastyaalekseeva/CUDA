#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
//����
__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}
//������� �������
int main()
{
	
	int a, b, c; // ���������� �� CPU
	int *dev_a, *dev_b, *dev_c; // ���������� �� GPU
	int size = sizeof(int); //�����������
							//�������� ������ �� GPU
	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);
	//������������� ����������
	a = 2;
	b = 7;
	// ����������� ���������� � CPU �� GPU
	hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);
	hipEvent_t start, stop;//�������������� �������
	float elapsedTime;
	hipEventCreate(&start);//������� �������
	hipEventCreate(&stop);
	hipEventRecord(start, 0);//������ ������� 
	// ����� ����
	add <<< 1, 1 >>>(dev_a, dev_b, dev_c);
	//����������� ���������� ������ ���� � GPU �� CPU
	hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);
	//�������� ������ �� GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);//�������� ���������� ������ ����
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time spent executing by the GPU: %.2f millseconds\n", elapsedTime
		);//����� ����������
	hipEventDestroy(start);//����������� �������
	hipEventDestroy(stop);
	system("pause");
}