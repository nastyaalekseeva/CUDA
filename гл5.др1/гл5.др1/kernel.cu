#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h> 
#include <math.h>
__global__ void a_f(float *A, float *x,float *y)
{
		*A = __fadd_rz(*x,*y);
}

__global__ void mul_f(float *A, float *x, float *y)
{
		*A = __fmul_rz(*x, *y);
}
__global__ void sqrt_f(float *A, float *x)
{
		*A = __fsqrt_rz(*x);
}
__global__ void a_d(double *A, double *x, double *y)
{
		*A = __dadd_rz(*x, *y);
}

__global__ void mul_d(double *A, double *x, double *y)
{
		*A = __dmul_rz(*x, *y);
}
__global__ void sqrt_d(double *A, double *x)
{
		*A = __dsqrt_rz(*x);
}
int main()
{
	float x=5.1553250, S[1], y=4.5467960;
	float *A, *dev_x, *dev_y;
	double arr[1],xd = 5.155325, yd = 4.546796;
	double *A_d, *dev_xd, *dev_yd;

	int size = sizeof(float);
	hipMalloc((void**)&A, size);
	hipMalloc((void**)&dev_x, size);
	hipMemcpy(dev_x, &x, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_y, size);
	hipMemcpy(dev_y, &y, size, hipMemcpyHostToDevice);
	hipEvent_t start, stop; 
	float elapsedTime;
	hipEventCreate(&start); 
	hipEventCreate(&stop);  
	hipEventRecord(start, 0); 
	a_f << <1, 1 >> > (A, dev_x,dev_y);
	//mul_f << <1, 1 >> > (A, dev_x, dev_y);
	//sqrt_f << <1, 1 >> > (A, dev_x);
	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipEventSynchronize(stop); 
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time spent executing by the GPU: %.2f millseconds\n", elapsedTime); 
	hipEventDestroy(start); 
	hipEventDestroy(stop);
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("GPUassert: %s %s %d\n", hipGetErrorString(err));
	}
	hipMemcpy(S, A, size, hipMemcpyDeviceToHost);

	int size_d = sizeof(double);
	hipMalloc((void**)&A_d, size_d);
	hipMalloc((void**)&dev_xd, size_d);
	hipMemcpy(dev_xd, &xd, size_d, hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_yd, size_d);
	hipMemcpy(dev_yd, &yd, size_d, hipMemcpyHostToDevice);
	hipEvent_t start1, stop1;
	float elapsedTime1; 
	hipEventCreate(&start1);
	hipEventCreate(&stop1);  
	hipEventRecord(start1, 0);
	a_d << <1, 1 >> > (A_d, dev_xd, dev_yd);
	//mul_d << <1, 1 >> > (A_d, dev_xd, dev_yd);
	//sqrt_d << <1, 1 >> > (A_d, dev_xd);
	hipEventRecord(stop1, 0);
	hipDeviceSynchronize();
	hipEventSynchronize(stop1); 
	hipEventElapsedTime(&elapsedTime1, start1, stop1); 
	printf("Time spent executing by the GPU: %.2f millseconds\n", elapsedTime1); 
	hipEventDestroy(start1);
	hipEventDestroy(stop1);
	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
	{
		printf("GPUassert: %s %s %d\n", hipGetErrorString(err1));
	}
	hipMemcpy(arr, A_d, size_d, hipMemcpyDeviceToHost);

		printf("x=%f, y=%f,  S=%f,  arr=%f \n", x,y, S[0], arr[0]);

	getchar();
}