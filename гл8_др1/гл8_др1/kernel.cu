#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cmath>
#include <assert.h>
#define  NUM_THREADS 100
//#define N 10000
texture<float, 1, hipReadModeElementType> texRef1;
texture<float, 1, hipReadModeElementType> texRef2;
texture<float, 1, hipReadModeElementType> texRef3;
texture<float, 1, hipReadModeElementType> texRef4;
__global__ void scalMult(float *C, int N)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < N)
	{
		float x = tex1D(texRef1, float(idx));
		float y= tex1D(texRef2, float(idx));
		C[idx] = x*y;
	}
}

__global__ void scalMult1(float *C, int N)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < N)
	{
		float y = tex1Dfetch(texRef3, idx);
		float x = tex1D(texRef1, float(idx));
		//printf("j=%d x=%f, y=%f \n", idx, x, y);
		C[idx] = x*y;
	}
}
__global__ void scalMult2(float *C, int N)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < N)
	{
		float y = tex1Dfetch(texRef3, idx);
		float x = tex1Dfetch(texRef4, idx);
		//printf("j=%d x=%f, y=%f \n", idx, x, y);
		C[idx] = x*y;
	}
}
void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
int main()
{
	float *dev_S;
	float *h_S;
	for (int N = 1000; N <= 50000; N = N + 1000)
	{
		int nBlocks = N / NUM_THREADS;
		printf("N=%d\n",N);
		hipMalloc((void**)&dev_S, sizeof(float)*N);
		h_S= (float*)malloc(sizeof(float)*N);
		float *x = (float*)malloc(N * sizeof(float));
		float *y = (float*)malloc(N * sizeof(float));
		for (int i = 0; i < N; i++)
		{
			x[i] = 2;
			y[i] = 1;
		}
		hipArray* cuArray_x;
		hipMallocArray(&cuArray_x, &texRef1.channelDesc, N, 1);
		hipMemcpyToArray(cuArray_x, 0, 0, x, sizeof(float)*N, hipMemcpyHostToDevice);
		hipBindTextureToArray(texRef1, cuArray_x);
		hipArray* cuArray_y;
		hipMallocArray(&cuArray_y, &texRef2.channelDesc, N, 1);
		hipMemcpyToArray(cuArray_y, 0, 0, y, sizeof(float)*N, hipMemcpyHostToDevice);
		hipBindTextureToArray(texRef2, cuArray_y);
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		scalMult << <nBlocks, NUM_THREADS >> > (dev_S, N);
		hipEventRecord(stop, 0); hipEventSynchronize(stop);
		float KernelTime;
		hipEventElapsedTime(&KernelTime, start, stop);
		printf("1. CUDA Array\n");
		printf("KernelTime: %.2f milliseconds\n", KernelTime);
		hipMemcpy(h_S, dev_S, sizeof(float)*N, hipMemcpyDeviceToHost);
		float sum = 0.0;
		for (int i = 0; i < N; i++)
		{
			sum = sum + h_S[i];
		}
		printf("GPU: %f \n", sum);
		float sum1 = 0.0;
		for (int i = 0; i < N; i++)
		{
			sum1 = sum1 + x[i] * y[i];
		}
		printf("CPU: %f \n", sum1);

		float *d_y1;
		int memSize = N * sizeof(float);
		hipMalloc((void**)&d_y1, memSize);
		hipMemcpy(d_y1, y, memSize, hipMemcpyHostToDevice);
		hipBindTexture(0, texRef3, d_y1, memSize);
		checkCUDAError("bind");
		hipEvent_t start1, stop1;
		hipEventCreate(&start1);
		hipEventCreate(&stop1);
		hipEventRecord(start1, 0);
		scalMult1 << <nBlocks, NUM_THREADS >> > (dev_S, N);
		hipDeviceSynchronize();
		hipEventRecord(stop1, 0); 
		hipEventSynchronize(stop1);
		float KernelTime1;
		hipEventElapsedTime(&KernelTime1, start1, stop1);
		printf("2. CUDA Array and linear memory \n");
		printf("KernelTime: %.2f milliseconds\n", KernelTime1);
		hipMemcpy(h_S, dev_S, memSize, hipMemcpyDeviceToHost);
		sum = 0.0;
		for (int i = 0; i < N; i++)
		{
			sum = sum + h_S[i];
		}
		printf("GPU: %f \n", sum);
		sum1 = 0.0;
		for (int i = 0; i < N; i++)
		{
			sum1 = sum1 + x[i] * y[i];
		}
		printf("CPU: %f \n", sum1);


		hipMalloc((void**)&d_y1, memSize);
		hipMemcpy(d_y1, y, memSize, hipMemcpyHostToDevice);
		hipBindTexture(0, texRef3, d_y1, memSize);
		float *d_x1;
		hipMalloc((void**)&d_x1, memSize);
		hipMemcpy(d_x1, x, memSize, hipMemcpyHostToDevice);
		hipBindTexture(0, texRef4, d_x1, memSize);
		checkCUDAError("bind");
		hipEvent_t start2, stop2;
		hipEventCreate(&start2);
		hipEventCreate(&stop2);
		hipEventRecord(start2, 0);
		scalMult2 << <nBlocks, NUM_THREADS >> > (dev_S, N);
		hipDeviceSynchronize();
		hipEventRecord(stop2, 0);
		hipEventSynchronize(stop2);
		float KernelTime2;
		hipEventElapsedTime(&KernelTime2, start2, stop2);
		printf("3. Linear memory \n");
		printf("KernelTime: %.2f milliseconds\n", KernelTime2);
		hipMemcpy(h_S, dev_S, memSize, hipMemcpyDeviceToHost);
		sum = 0.0;
		for (int i = 0; i < N; i++)
		{
			sum = sum + h_S[i];
		}
		printf("GPU: %f \n", sum);
		sum1 = 0.0;
		for (int i = 0; i < N; i++)
		{
			sum1 = sum1 + x[i] * y[i];
		}
		printf("CPU: %f \n", sum1);
		printf("\n");
	}
	free(h_S);
	hipFree(dev_S);
	hipUnbindTexture(texRef1);
	hipUnbindTexture(texRef2);
	hipUnbindTexture(texRef3);
	hipUnbindTexture(texRef4);
	getchar();
}