#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define  n 100000
#define BLOCK 1000
__global__ void MK(int *S, unsigned int seed)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	float x, y;
	hiprandState_t m;
	hiprand_init(seed, i, 0, &m);
	x = hiprand_uniform(&m);
	hiprand_init(NULL, i, 0, &m);
	y =hiprand_uniform(&m);
	//printf("x=%f, y=%f \n", x,y);
	if (x*x + y*y <= 1)
		atomicAdd(S, 1);
	
}
void main()
{
	int host_S;
	int *dev_S;
	int size = sizeof(int);
	hipMalloc((void**)&dev_S, size);
	MK <<< BLOCK,n/BLOCK >>>(dev_S, time(NULL));
	hipDeviceSynchronize();
	hipMemcpy(&host_S, dev_S, size, hipMemcpyDeviceToHost);
	hipFree(dev_S);
	printf("pi=%f", (4 * host_S*1.0)/n);

	system("pause");
}
