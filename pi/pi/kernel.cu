
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <cmath>
#define  n 10000
#define BLOCK 1000
 __global__ void Su(float *a,float *b,float *h)
 {
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	b[i] = (*h)*sqrtf(1 - a[i] * a[i]);
 }
int main()
{
	float a = 0, b = 1;
	float h = ((b - a)*1.0) / n;
	float x[n],S[n];//CPU
	float *dev_S,*dev_x,*dev_h;//GPU
	x[0] = a;
	for (int i = 1; i < n; i++)
	{
		x[i] = i*h;
	}
	int size = sizeof(float);
	hipMalloc((void**)&dev_S, n*size);
	hipMalloc((void**)&dev_x, n*size);
	hipMalloc((void**)&dev_h, size);
	hipMemcpy(dev_x, &x,n*size, hipMemcpyHostToDevice);
	hipMemcpy(dev_h, &h, size, hipMemcpyHostToDevice);
	Su<<< BLOCK,n/BLOCK >>>(dev_x,dev_S,dev_h);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf( "GPUassert: %s %s %d\n", hipGetErrorString(err));
	}
	hipMemcpy(&S, dev_S, n*size, hipMemcpyDeviceToHost);
	float p = 0.0;
	for (int i = 0; i < n; i++)
	{
		p=p+S[i];
	}
	hipFree(dev_S);
	printf("Pi = %f ", 4*p);
	getchar();
}
